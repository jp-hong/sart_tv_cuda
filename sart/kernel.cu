#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <cstdlib>
#include <cmath>

#include "parameters.h"
#include "util.h"
#include "cuda_sart.cuh"
#include "cuda_tv.cuh"


void loadImage(float *d_image1D, int nBytes, int len, char *fileName);

void loadImagePinnedMem(float *h_data, float *d_data, int nBytes, int len, char *fileName);

void saveImage(float *d_data1D, int len, int nBytes, char *fileName);

inline float getAngle(int view)
{
	return S0 + view * SLEN / NS;
}

inline void startGPUTimer(hipEvent_t start)
{
	HANDLE_ERROR(hipEventRecord(start));
}

inline void stopGPUTimer(hipEvent_t stop)
{
	HANDLE_ERROR(hipEventRecord(stop));
	HANDLE_ERROR(hipEventSynchronize(stop));
}

inline float getElapsedTime(hipEvent_t start, hipEvent_t stop)
{
	float ms;
	HANDLE_ERROR(hipEventElapsedTime(&ms, start, stop));
	return ms;
}

int main()
{
	//GPU timer
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	float ms, totalTime;
	
	//1D block for general use
	dim3 block(256);
	dim3 grid(0);

	//2D block for detector array (forward projection)
	dim3 block2d(32, 16);
	dim3 grid2d(iDivUp(NU, block2d.x), iDivUp(NV, block2d.y));

	//3D block for image volume (back projection)
	dim3 block3d(8, 8, 8);
	dim3 grid3d(iDivUp(NX, block3d.x), iDivUp(NY, block3d.y), iDivUp(NZ, block3d.z));

	//All memory allocation
	char *fileName = (char *)malloc(sizeof(char) * 512);

	float *h_data, *ind, *data, *proj, *corImage, *image, *diffImage, *normImage, *prevImage;

	HANDLE_ERROR(hipMalloc((void **)&data, PROJ_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&proj, PROJ_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&corImage, PROJ_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&image, IMAGE_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&diffImage, IMAGE_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&normImage, IMAGE_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&prevImage, IMAGE_BYTES));
	HANDLE_ERROR(hipHostMalloc((void **)&h_data, PROJ_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&ind, PROJ_BYTES));

	//CUDA array for 3D texture
	const hipExtent volumeSize = make_hipExtent(NX, NY, NZ);
	createCudaArray(volumeSize);

	float angle, cosbeta, sinbeta;
	int deltaBeta = 1;

	float centernBins = (NU - 1.0) / 2.0;
	float ScaleFactor = R / D;
	float deltaS = DU*ScaleFactor;
	float centernZZ = (NV - 1.0) / 2.0;
	float deltaZZ0 = DV;
	float deltaZZ = deltaZZ0*ScaleFactor;

	float lamda = LAMDA, gf = GF, adist, ddist, gdist;

	//normImage generation
	printf("Generating normalization image ... ");

	startGPUTimer(start);

	getGridDim(&grid, block, IMAGE_LEN);
	setArrVal <<<grid, block>>> (image, IMAGE_LEN, 1);
	bind3DTexture(image, volumeSize);
	HANDLE_ERROR(hipMemset(normImage, 0, IMAGE_BYTES));

	for (int i = 0; i < NS; i += N_SKIP)
	{
		angle = getAngle(i);
		cosbeta = cosf(angle);
		sinbeta = sinf(angle);
		
		forwardProjectionTexKernel <<<grid2d, block2d>>> (proj, sinbeta, cosbeta, 
			R, D, NU, NV, DU, DV, U0, V0, DX, DY, DZ, X0, Y0, Z0, NX, NY, NZ, XLEN, YLEN, ZLEN);

		backProjectionNormKernel <<<grid3d, block3d>>> (normImage, proj, R, cosbeta, sinbeta, 
			deltaZZ, centernZZ, deltaS, centernBins, NV, NU, deltaBeta, NX, NY, NZ, DX, DY, DZ);
	}

	stopGPUTimer(stop);
	ms = getElapsedTime(start, stop);
	printf("Elapsed time : %.3f ms\n", ms);

	//reconstruction
	printf("\n --- STARTING RECONSTRUCTION WITH SART ---\n");

	HANDLE_ERROR(hipMemset(image, 0, IMAGE_BYTES));
	HANDLE_ERROR(hipMemset(diffImage, 0, IMAGE_BYTES));

	for (int i = 0; i < N_ITER; i++)
	{
		totalTime = 0;

		printf("\nIteration : %03d\n", i + 1);
		startGPUTimer(start);

		bind3DTexture(image, volumeSize);

		if (USE_TV && i >= 10)
			HANDLE_ERROR(hipMemcpy(prevImage, image, IMAGE_BYTES, hipMemcpyDeviceToDevice));

		//SART step
		printf("\tSART step          ");
		for (int j = 0; j < NS; j += N_SKIP)
		{
			angle = getAngle(j);
			cosbeta = cosf(angle);
			sinbeta = sinf(angle);
			
			generateFileName(fileName, INPUT_DIR, "", j, ".dat");
			loadImagePinnedMem(h_data, data, PROJ_BYTES, PROJ_LEN, fileName);

			forwardProjectionTexKernel <<<grid2d, block2d>>> (proj, sinbeta, cosbeta, 
				R, D, NU, NV, DU, DV, U0, V0, DX, DY, DZ, X0, Y0, Z0, NX, NY, NZ, XLEN, YLEN, ZLEN);

			dataFlipU <<<grid2d, block2d>>> (data, NU, NV);

			getGridDim(&grid, block, PROJ_LEN);
			correctiveImageKernel <<<grid, block>>> (data, proj, corImage, PROJ_LEN);

			backProjectionKernel <<<grid3d, block3d>>> (diffImage, corImage, R, cosbeta, 
				sinbeta, deltaZZ, centernZZ, deltaS, centernBins, NV, NU, deltaBeta, normImage, 
				NX, NY, NZ, DX, DY, DZ);

			getGridDim(&grid, block, IMAGE_LEN);
			nanAndInfCheck <<<grid, block>>> (diffImage, IMAGE_LEN);
		}
		
		imageUpdatekernel <<<grid, block>>> (image, diffImage, IMAGE_LEN, lamda);
		zeroOutsideFOV <<<grid3d, block3d>>> (image, NX, NY, NZ);
		makePositive <<<grid, block>>> (image, IMAGE_LEN);
		HANDLE_ERROR(hipMemset(diffImage, 0, IMAGE_BYTES));

		stopGPUTimer(stop);
		ms = getElapsedTime(start, stop);
		totalTime += ms;
		printf("Elapsed time : %.3f ms\n", ms);

		//TV step
		if (USE_TV && i >= 10)
		{
			printf("\tGet data distance  ");
			startGPUTimer(start);

			bind3DTexture(image, volumeSize);

			getGridDim(&grid, block, IMAGE_LEN);
			subtractionKernel <<<grid, block>>> (diffImage, image, prevImage, IMAGE_LEN);
			HANDLE_ERROR(hipDeviceSynchronize());
			adist = sqrtf(sumSquaresGPU(diffImage, IMAGE_LEN));

			if (i == 10)
				gf *= adist;

			getGridDim(&grid, block, PROJ_LEN);
			ddist = 0;

			for (int j = 0; j < NS; j += N_SKIP)
			{
				generateFileName(fileName, INPUT_DIR, "", j, ".dat");
				loadImagePinnedMem(h_data, data, PROJ_BYTES, PROJ_LEN, fileName);

				forwardProjectionTexKernel <<<grid2d, block2d>>> (proj, sinbeta, cosbeta,
					R, D, NU, NV, DU, DV, U0, V0, DX, DY, DZ, X0, Y0, Z0, NX, NY, NZ, XLEN, YLEN, ZLEN);

				dataFlipU <<<grid2d, block2d>>> (data, NU, NV);

				correctiveImageKernel <<<grid, block>>> (data, proj, corImage, PROJ_LEN);

				ddist += sumSquaresGPU(corImage, PROJ_LEN);
			}

			ddist = sqrtf(ddist);

			stopGPUTimer(stop);
			ms = getElapsedTime(start, stop);
			totalTime += ms;
			printf("Elapsed time : %.3f ms\n", ms);

			HANDLE_ERROR(hipMemcpy(prevImage, image, IMAGE_BYTES, hipMemcpyDeviceToDevice));

			getGridDim(&grid, block, IMAGE_LEN);
			for (int j = 0; j < N_TV; j++)
			{
				printf("\tTV step %02d         ", j + 1);
				startGPUTimer(start);

				TVStep(image, NX, NY, NZ, gf);
				makePositive <<<grid, block>>> (image, IMAGE_LEN);

				stopGPUTimer(stop);
				ms = getElapsedTime(start, stop);
				totalTime += ms;
				printf("Elapsed time : %.3f ms\n", ms);
			}

			subtractionKernel <<<grid, block>>> (diffImage, image, prevImage, IMAGE_LEN);
			gdist = sqrtf(sumSquaresGPU(diffImage, IMAGE_LEN));

			if (gdist >= CONV * adist && ddist > MEPS)
				gf *= RED_FACT;
		}

		lamda *= RED_REG;

		printf("\n\tTotal iteration time : %.3f ms\n", totalTime);

		//save image
		if (i == 0 || (i + 1) % SAVE_INTERVAL == 0)
		{
			generateFileName(fileName, OUTPUT_DIR, SAVE_FILE_NAME, i + 1, ".dat");
			saveImage(image, IMAGE_LEN, IMAGE_BYTES, fileName);
			printf("\tSaved file \"%s\"\n", fileName);
		}
	}

	printf("\n --- RECONSTRUCTION FINISHED AFTER %d ITERATIONS ---\n\n", N_ITER);

	//Free all allocated memory
	free3DTexture();
	free(fileName);
	HANDLE_ERROR(hipFree(data));
	HANDLE_ERROR(hipFree(proj));
	HANDLE_ERROR(hipFree(corImage));
	HANDLE_ERROR(hipFree(image));
	HANDLE_ERROR(hipFree(diffImage));
	HANDLE_ERROR(hipFree(normImage));
	HANDLE_ERROR(hipFree(prevImage));
	HANDLE_ERROR(hipFree(ind));
	HANDLE_ERROR(hipHostFree(h_data));
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	//Reset device state
	HANDLE_ERROR(hipDeviceReset());

	return 0;
}

void loadImage(float *d_image1D, int nBytes, int len, char *fileName)
{
	float *h_image1D = (float*)malloc(nBytes);
	readArrayFromFile(h_image1D, len, fileName, sizeof(float));
	HANDLE_ERROR(hipMemcpy(d_image1D, h_image1D, nBytes, hipMemcpyHostToDevice));
	free(h_image1D);
}

void loadImagePinnedMem(float *h_data, float *d_data, int nBytes, int len, char *fileName)
{
	readArrayFromFile(h_data, PROJ_LEN, fileName, sizeof(float));
	HANDLE_ERROR(hipMemcpy(d_data, h_data, PROJ_BYTES, hipMemcpyHostToDevice));
}

void saveImage(float *d_data1D, int len, int nBytes, char *fileName)
{
	float *h_data1D = (float*)malloc(nBytes);
	HANDLE_ERROR(hipMemcpy(h_data1D, d_data1D, nBytes, hipMemcpyDeviceToHost));
	writeArrayToFile(h_data1D, len, fileName, sizeof(float));
	free(h_data1D);
}